#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "kmeans.h"
#include "alloc.h"
#include "error.h"

#ifdef __HIPCC__
inline void checkCuda(hipError_t e) {
    if (e != hipSuccess) {
        // hipGetErrorString() isn't always very helpful. Look up the error
        // number in the hipError_t enum in hip/driver_types.h in the CUDA includes
        // directory for a better explanation.
        error("CUDA Error %d: %s\n", e, hipGetErrorString(e));
    }
}

inline void checkLastCudaError() {
    checkCuda(hipGetLastError());
}
#endif

__device__ int get_tid(){
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	return tid;
}

/* square of Euclid distance between two multi-dimensional points using column-base format */
__host__ __device__ inline static
float euclid_dist_2_transpose(int numCoords,
                    int    numObjs,
                    int    numClusters,
                    float *objects,     // [numCoords][numObjs]
                    float *clusters,    // [numCoords][numClusters]
                    int    objectId,
                    int    clusterId)
{
    int i;
    float ans=0.0;
    float coord1, coord2;

    for (i = 0; i < numCoords; i++){
        coord1 = objects[i * numObjs + objectId];
        coord2 = clusters[i * numClusters + clusterId];
        ans += (coord1 - coord2) * (coord1 - coord2);
    }
    return(ans);
}

__global__ static
void find_nearest_cluster(int numCoords,
                          int numObjs,
                          int numClusters,
                          float *deviceobjects,           //  [numCoords][numObjs]
/*                          
                          TODO: If you choose to do (some of) the new centroid calculation here, you will need some extra parameters here (from "update_centroids").
*/                          
                          float *deviceClusters,    //  [numCoords][numClusters]
                          int *deviceMembership,          //  [numObjs]
                          float *devdelta)
{
    extern __shared__ float shmemClusters[];

	/* TODO: Copy deviceClusters to shmemClusters so they can be accessed faster. 
		BEWARE: Make sure operations is complete before any thread continues... */
    int index, i, j;

    //Grid-stride loop for moving data
    for (j = threadIdx.x; j < numClusters * numCoords; j += blockDim.x)
        shmemClusters[j] = deviceClusters[j];

    __syncthreads();

	/* Get the global ID of the thread. */
    int tid = get_tid(); 

    // Bounds check
    if (tid >= numObjs) return;

    
    float dist, min_dist;
    /* find the cluster id that has min distance to object */
    index = 0;

    /* TODO: call min_dist = euclid_dist_2(...) with correct objectId/clusterId using clusters in shmem*/
    min_dist = euclid_dist_2_transpose(numCoords, numObjs, numClusters, objects, shmemClusters, tid, 0);
    for (i=1; i<numClusters; i++) {
        /* TODO: call dist = euclid_dist_2(...) with correct objectId/clusterId using clusters in shmem*/
        dist = euclid_dist_2_transpose(numCoords, numObjs, numClusters, objects, shmemClusters, tid, i);

        /* no need square root */
        if (dist < min_dist) { /* find the min and its array index */
            min_dist = dist;
            index    = i;
        }
    }
    if (deviceMembership[tid] != index) {
        atomicAdd(devdelta, 1.0);
    }
    /* assign the deviceMembership to object objectId */
    deviceMembership[tid] = index;
}

__global__ static
void update_centroids(int numCoords,
                          int numObjs,
                          int numClusters,
                          int *devicenewClusterSize,           //  [numClusters]
                          float *devicenewClusters,    //  [numCoords][numClusters]
                          float *deviceClusters,
                          float *deviceClusters,    //  [numCoords][numClusters]
                          int *deviceMembership
                          )    //  [numCoords][numClusters])
{

    extern __shared__ int shmemMembership[];
    /* TODO: additional steps for calculating new centroids in GPU? */
    int index;
    int tid = get_tid();

    if (idx >= numObjs) return;

    index = deviceMembership[tid];


    
}

//
//  ----------------------------------------
//  DATA LAYOUT
//
//  objects         [numObjs][numCoords]
//  clusters        [numClusters][numCoords]
//  dimObjects      [numCoords][numObjs]
//  dimClusters     [numCoords][numClusters]
//  newClusters     [numCoords][numClusters]
//  deviceObjects   [numCoords][numObjs]
//  deviceClusters  [numCoords][numClusters]
//  ----------------------------------------
//
/* return an array of cluster centers of size [numClusters][numCoords]       */            
void kmeans_gpu(	float *objects,      /* in: [numObjs][numCoords] */
		               	int     numCoords,    /* no. features */
		               	int     numObjs,      /* no. objects */
		               	int     numClusters,  /* no. clusters */
		               	float   threshold,    /* % objects change membership */
		               	long    loop_threshold,   /* maximum number of iterations */
		               	int    *membership,   /* out: [numObjs] */
						float * clusters,   /* out: [numClusters][numCoords] */
						int blockSize)  
{
    double timing = wtime(), timing_internal, timer_min = 1e42, timer_max = 0;

    double cpu_timing = 0, gpu_timing = 0, transfer_timing_before = 0, transfer_timing_after= 0;
    double cpu_total_time = 0, gpu_total_time = 0, transfer_total_time = 0; 

	int    loop_iterations = 0; 
    int      i, j, index, loop=0;
    float  delta = 0, *dev_delta_ptr;          /* % of objects change their clusters */
    /* TODO: Copy me from transpose version*/
    float  **dimObjects = NULL; //calloc_2d(...) -> [numCoords][numObjs]
    float  **dimClusters = NULL;  //calloc_2d(...) -> [numCoords][numClusters]
    float  **newClusters = NULL;  //calloc_2d(...) -> [numCoords][numClusters]

    double cpu_start, cpu_end, gpu_start, gpu_end, cpu_gpu_transfer_start, cpu_gpu_transfer_end;

    printf("\n|-----------Full-offload GPU Kmeans------------|\n\n");
    
    /* TODO: Copy me from transpose version*/
	for(i = 0; i < numObjs; i++)
        for(j = 0; j < numCoords; j++)
            dimObjects[j][i] = objects[i*numCoords + j];
    
    float *deviceObjects;
    float *deviceClusters, *devicenewClusters;
    int *deviceMembership;
    int *devicenewClusterSize; /* [numClusters]: no. objects assigned in each new cluster */
    
    /* pick first numClusters elements of objects[] as initial cluster centers*/
    for (i = 0; i < numCoords; i++) {
        for (j = 0; j < numClusters; j++) {
            dimClusters[i][j] = dimObjects[i][j];
        }
    }
	
    /* initialize membership[] */
    for (i=0; i<numObjs; i++) membership[i] = -1;
    
    timing = wtime() - timing;
    printf("t_alloc: %lf ms\n\n", 1000*timing);
    timing = wtime(); 
    const unsigned int numThreadsPerClusterBlock = (numObjs > blockSize)? blockSize: numObjs;
    const unsigned int numClusterBlocks = (numObjs + numThreadsPerClusterBlock - 1) / numThreadsPerClusterBlock; 

	/*	Define the shared memory needed per block.
    	- BEWARE: We can overrun our shared memory here if there are too many
    	clusters or too many coordinates! 
    	- This can lead to occupancy problems or even inability to run. 
    	- Your exercise implementation is not requested to account for that (e.g. always assume deviceClusters fit in shmemClusters */
    const unsigned int clusterBlockSharedDataSize = numCoords * numClusters * sizeof(float); 
    const unsigned int membershipBlockSharedDataSize = numObjs * sizeof(int); 

    hipDeviceProp_t deviceProp;
    int deviceNum;
    hipGetDevice(&deviceNum);
    hipGetDeviceProperties(&deviceProp, deviceNum);

    if (clusterBlockSharedDataSize > deviceProp.sharedMemPerBlock) {
        error("Your CUDA hardware has insufficient block shared memory to hold all cluster centroids\n");
    }
           
    checkCuda(hipMalloc(&deviceObjects, numObjs*numCoords*sizeof(float)));
    checkCuda(hipMalloc(&deviceClusters, numClusters*numCoords*sizeof(float)));
    checkCuda(hipMalloc(&devicenewClusters, numClusters*numCoords*sizeof(float)));
    checkCuda(hipMalloc(&devicenewClusterSize, numClusters*sizeof(int)));
    checkCuda(hipMalloc(&deviceMembership, numObjs*sizeof(int)));
    checkCuda(hipMalloc(&dev_delta_ptr, sizeof(float)));
 
    timing = wtime() - timing;
    printf("t_alloc_gpu: %lf ms\n\n", 1000*timing);
    timing = wtime(); 
       
    checkCuda(hipMemcpy(deviceObjects, dimObjects[0],
              numObjs*numCoords*sizeof(float), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(deviceMembership, membership,
              numObjs*sizeof(int), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(deviceClusters, dimClusters[0],
                  numClusters*numCoords*sizeof(float), hipMemcpyHostToDevice));
    checkCuda(hipMemset(devicenewClusterSize, 0, numClusters*sizeof(int)));
    free(dimObjects[0]);
      
    timing = wtime() - timing;
    printf("t_get_gpu: %lf ms\n\n", 1000*timing);
    timing = wtime();   
    
    do {
        timing_internal = wtime(); 
        checkCuda(hipMemset(dev_delta_ptr, 0, sizeof(float)));          
		//printf("Launching find_nearest_cluster Kernel with grid_size = %d, block_size = %d, shared_mem = %d KB\n", numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize/1000);
        /* TODO: change invocation if extra parameters needed */
        find_nearest_cluster
            <<< numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize >>>
            (numCoords, numObjs, numClusters,
             deviceObjects, devicenewClusterSize, devicenewClusters, deviceClusters, deviceMembership, dev_delta_ptr);

        hipDeviceSynchronize(); checkLastCudaError();
    
    	/* TODO: Copy dev_delta_ptr to &delta */
        checkCuda(hipMemcpy(&delta, dev_delta_ptr, sizeof(float), hipMemcpyDeviceToHost));

     	const unsigned int update_centroids_block_sz = (numCoords* numClusters > blockSize) ? blockSize: numCoords* numClusters;  /* TODO: can use different blocksize here if deemed better */
     	const unsigned int update_centroids_dim_sz =  -1; /* TODO: calculate dim for "update_centroids" and fire it 
     	update_centroids<<< update_centroids_dim_sz, update_centroids_block_sz, 0 >>>
            (numCoords, numClusters, devicenewClusterSize, devicenewClusters, deviceClusters);  */  
        hipDeviceSynchronize(); checkLastCudaError();   
                       
        delta /= numObjs;
       	//printf("delta is %f - ", delta);
        loop++; 
        //printf("completed loop %d\n", loop);
		timing_internal = wtime() - timing_internal; 
		if ( timing_internal < timer_min) timer_min = timing_internal; 
		if ( timing_internal > timer_max) timer_max = timing_internal; 
	} while (delta > threshold && loop < loop_threshold);
                  	
    checkCuda(hipMemcpy(membership, deviceMembership,
                 numObjs*sizeof(int), hipMemcpyDeviceToHost));     
    checkCuda(hipMemcpy(dimClusters[0], deviceClusters,
                 numClusters*numCoords*sizeof(float), hipMemcpyDeviceToHost));  
                                   
	for (i=0; i<numClusters; i++) {
		for (j=0; j<numCoords; j++) {
		    clusters[i*numCoords + j] = dimClusters[j][i];
		}
	}
	
    timing = wtime() - timing;
    printf("nloops = %d  : total = %lf ms\n\t-> t_loop_avg = %lf ms\n\t-> t_loop_min = %lf ms\n\t-> t_loop_max = %lf ms\n\n|-------------------------------------------|\n", 
    	loop, 1000*timing, 1000*timing/loop, 1000*timer_min, 1000*timer_max);

	char outfile_name[1024] = {0}; 
	sprintf(outfile_name, "Execution_logs/Sz-%ld_Coo-%d_Cl-%d.csv", numObjs*numCoords*sizeof(float)/(1024*1024), numCoords, numClusters);
	FILE* fp = fopen(outfile_name, "a+");
	if(!fp) error("Filename %s did not open succesfully, no logging performed\n", outfile_name); 
	fprintf(fp, "%s,%d,%lf,%lf,%lf\n", "All_GPU", blockSize, timing/loop, timer_min, timer_max);
	fclose(fp); 
	
    checkCuda(hipFree(deviceObjects));
    checkCuda(hipFree(deviceClusters));
    checkCuda(hipFree(devicenewClusters));
    checkCuda(hipFree(devicenewClusterSize));
    checkCuda(hipFree(deviceMembership));

    return;
}

